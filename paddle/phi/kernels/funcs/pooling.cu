#include "hip/hip_runtime.h"
/* Copyright (c) 2022 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>

#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/fast_divmod.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/kernels/funcs/pooling.h"

namespace phi {
namespace funcs {

struct FastDivModForPooling {
 public:
  paddle::platform::FastDivMod channel;
  paddle::platform::FastDivMod width;
  paddle::platform::FastDivMod height;

  explicit HOSTDEVICE FastDivModForPooling(const int channels,
                                           const int output_width,
                                           const int output_height) {
    channel = paddle::platform::FastDivMod(channels);
    width = paddle::platform::FastDivMod(output_width);
    height = paddle::platform::FastDivMod(output_height);
  }
};

struct FastDivModForPooling3D {
public:
 paddle::platform::FastDivMod channel;
 paddle::platform::FastDivMod width;
 paddle::platform::FastDivMod height;
 paddle::platform::FastDivMod depth;

 explicit HOSTDEVICE FastDivModForPooling3D(const int channels,
                                            const int output_width,
                                            const int output_height,
                                            const int output_depth) {
   channel = paddle::platform::FastDivMod(channels);
   width = paddle::platform::FastDivMod(output_width);
   height = paddle::platform::FastDivMod(output_height);
   depth = paddle::platform::FastDivMod(output_depth);
 }
};

struct FastDivModForPooling3DStride {
 public:
  paddle::platform::FastDivMod width;
  paddle::platform::FastDivMod height;
  paddle::platform::FastDivMod depth;

  explicit HOSTDEVICE FastDivModForPooling3DStride(const int stride_width,
                                                   const int stride_height,
                                                   const int stride_depth) {
    width = paddle::platform::FastDivMod(stride_width);
    height = paddle::platform::FastDivMod(stride_height);
    depth = paddle::platform::FastDivMod(stride_depth);                      
  }
};

struct FastDivModForPoolingWithMoreStaff {
 public:
  paddle::platform::FastDivMod channel;
  paddle::platform::FastDivMod width;
  paddle::platform::FastDivMod height;
  paddle::platform::FastDivMod ksize_w;
  paddle::platform::FastDivMod ksize_h;
  paddle::platform::FastDivMod stride_w;
  paddle::platform::FastDivMod stride_h;

  explicit HOSTDEVICE FastDivModForPoolingWithMoreStaff(
      const int channels,
      const int input_width,
      const int input_height,
      const int ksize_width,
      const int ksize_height,
      const int stride_width,
      const int stride_height) {
    channel = paddle::platform::FastDivMod(channels);
    width = paddle::platform::FastDivMod(input_width);
    height = paddle::platform::FastDivMod(input_height);
    ksize_w = paddle::platform::FastDivMod(ksize_width);
    ksize_h = paddle::platform::FastDivMod(ksize_height);
    stride_w = paddle::platform::FastDivMod(stride_width);
    stride_h = paddle::platform::FastDivMod(stride_height);
  }
};

struct FastDivModForPooling3DWithMoreStaff {
public:
 paddle::platform::FastDivMod channel;
 paddle::platform::FastDivMod width;
 paddle::platform::FastDivMod height;
 paddle::platform::FastDivMod depth;
 paddle::platform::FastDivMod ksize_w;
 paddle::platform::FastDivMod ksize_h;
 paddle::platform::FastDivMod ksize_d;
 paddle::platform::FastDivMod stride_w;
 paddle::platform::FastDivMod stride_h;
 paddle::platform::FastDivMod stride_d;

 explicit HOSTDEVICE FastDivModForPooling3DWithMoreStaff(
     const int channels,
     const int input_width,
     const int input_height,
     const int input_depth,
     const int ksize_width,
     const int ksize_height,
     const int ksize_depth,
     const int stride_width,
     const int stride_height,
     const int stride_depth
    ) {
   channel = paddle::platform::FastDivMod(channels);
   width = paddle::platform::FastDivMod(input_width);
   height = paddle::platform::FastDivMod(input_height);
   depth = paddle::platform::FastDivMod(input_depth);
   ksize_w = paddle::platform::FastDivMod(ksize_width);
   ksize_h = paddle::platform::FastDivMod(ksize_height);
   ksize_d = paddle::platform::FastDivMod(ksize_depth);
   stride_w = paddle::platform::FastDivMod(stride_width);
   stride_h = paddle::platform::FastDivMod(stride_height);
   stride_d = paddle::platform::FastDivMod(stride_depth);
 }
};

// 不过如果只优化我锚定的with_index kernel，就不需要写channel_last格式的索引，但KernelPool3D&KernelPool3DGrad&KernelMaxPool3DGrad是需要的，withindex的不管是2d还是3d其实都只支持一种格式
// 不过按照2d的标准是所有的kernel都有改写的，所以还是应该写一个通用的，写优化index的kernel，没问题以后再实现到所有3d kernel
template <typename FastDivModForPooling>
__device__ void OffsetPreparationFor4Dimension(int index,
                                               bool channel_last,
                                               FastDivModForPooling divmods,
                                               const int pad_width,
                                               const int pad_height,
                                               const int aux_width,
                                               const int aux_height,
                                               int* w_offset,
                                               int* h_offset,
                                               int* c_offset,
                                               int* stride) {
  if (!channel_last) { /* NCHW */
    auto input_width_divmod = divmods.width.Divmod(index);
    auto input_height_divmod = divmods.height.Divmod(input_width_divmod.val[0]);
    auto channel_divmod = divmods.channel.Divmod(input_height_divmod.val[0]);
    *w_offset = input_width_divmod.val[1] + pad_width;
    *h_offset = input_height_divmod.val[1] + pad_height;
    *c_offset = channel_divmod.val[1];
    *stride = (channel_divmod.val[0] * divmods.channel.divisor + *c_offset) *
              aux_height * aux_width;
  } else { /* NHWC */
    auto c_divmod = divmods.channel.Divmod(index);
    auto input_width_divmod = divmods.width.Divmod(c_divmod.val[0]);
    auto input_height_divmod = divmods.height.Divmod(input_width_divmod.val[0]);
    *c_offset = c_divmod.val[1];
    *w_offset = input_width_divmod.val[1] + pad_width;
    *h_offset = input_height_divmod.val[1] + pad_height;
    *stride = input_height_divmod.val[0] * aux_height * aux_width *
              divmods.channel.divisor;
  }
}

template <typename FastDivModForPooling3D>
__device__ void OffsetPreparationFor5Dimension(int index,
                                               bool channel_last,
                                               FastDivModForPooling3D divmods,
                                               const int pad_width,
                                               const int pad_height,
                                               const int pad_depth,
                                               const int aux_width,
                                               const int aux_height,
                                               const int aux_depth,
                                               int* w_offset,
                                               int* h_offset,
                                               int* d_offset,
                                               int* c_offset,
                                               int* stride) {
    if (!channel_last) { /* NCDHW */
      auto input_width_divmod = divmods.width.Divmod(index);
      auto input_height_divmod = divmods.height.Divmod(input_width_divmod.val[0]);
      auto input_depth_divmod = divmods.depth.Divmod(input_height_divmod.val[0]);
      auto channel_divmod = divmods.channel.Divmod(input_depth_divmod.val[0]);
      *w_offset = input_width_divmod.val[1] + pad_width;
      *h_offset = input_height_divmod.val[1] + pad_height;
      *d_offset = input_depth_divmod.val[1] + pad_depth;
      *c_offset = channel_divmod.val[1];
      *stride = (channel_divmod.val[0] * divmods.channel.divisor + *c_offset) * aux_depth * aux_height * aux_width;
    } else { /* NDHWC */
      auto channel_divmod = divmods.channel.Divmod(index);
      auto input_width_divmod = divmods.width.Divmod(channel_divmod.val[0]);
      auto input_height_divmod = divmods.height.Divmod(input_width_divmod.val[0]);
      auto input_depth_divmod = divmods.depth.Divmod(input_height_divmod.val[0]);
      *c_offset = channel_divmod.val[1];
      *w_offset = input_width_divmod.val[1] + pad_width;
      *h_offset = input_height_divmod.val[1] + pad_height;
      *d_offset = input_depth_divmod.val[1] + pad_depth;
      *stride = input_depth_divmod.val[0] * aux_depth * aux_height * aux_width * divmods.channel.divisor;
    }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool2D(const int nthreads,
                             const T* input_data,
                             const int channels,
                             const int input_height,
                             const int input_width,
                             const int output_height,
                             const int output_width,
                             const int ksize_height,
                             const int ksize_width,
                             const int stride_height,
                             const int stride_width,
                             const int padding_height,
                             const int padding_width,
                             FastDivModForPooling divmods,
                             PoolProcess pool_process,
                             bool exclusive,
                             bool adaptive,
                             T* output_data,
                             bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int hstart, hend, wstart, wend;
    int w_offset, h_offset, c_offset, input_offset;
    OffsetPreparationFor4Dimension<FastDivModForPooling>(index,
                                                         channel_last,
                                                         divmods,
                                                         0,
                                                         0,
                                                         input_width,
                                                         input_height,
                                                         &w_offset,
                                                         &h_offset,
                                                         &c_offset,
                                                         &input_offset);
    input_data += input_offset;

    if (adaptive) {
      hstart = AdaptStartIndex(h_offset, input_height, output_height);
      hend = AdaptEndIndex(h_offset, input_height, output_height);
      wstart = AdaptStartIndex(w_offset, input_width, output_width);
      wend = AdaptEndIndex(w_offset, input_width, output_width);
    } else {
      hstart = h_offset * stride_height - padding_height;
      hend = min(hstart + ksize_height, input_height);
      hstart = max(hstart, 0);
      wstart = w_offset * stride_width - padding_width;
      wend = min(wstart + ksize_width, input_width);
      wstart = max(wstart, 0);
    }

    T ele = pool_process.initial();
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        auto input_idx = channel_last
                             ? (h * input_width + w) * channels + c_offset
                             : h * input_width + w;
        pool_process.compute(input_data[input_idx], &ele);
      }
    }
    int pool_size = (exclusive || adaptive) ? (hend - hstart) * (wend - wstart)
                                            : ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}

template <typename T, typename PoolProcess>
__global__ void KernelPool2DGrad(const int nthreads,
                                 const T* __restrict__ input_data,
                                 const T* __restrict__ output_data,
                                 const T* __restrict__ output_grad,
                                 const int output_width,
                                 const int output_height,
                                 const int input_width,
                                 const int input_height,
                                 const int ksize_width,
                                 const int ksize_height,
                                 const int stride_width,
                                 const int stride_height,
                                 const int padding_width,
                                 const int padding_height,
                                 FastDivModForPoolingWithMoreStaff divmods,
                                 PoolProcess pool_process,
                                 bool exclusive,
                                 bool adaptive,
                                 T* __restrict__ input_grad,
                                 bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    T input = static_cast<T>(0);
    T input_grad_data = static_cast<T>(0);
    int phstart, phend, pwstart, pwend;
    int w_offset, h_offset, c_offset, output_offset;
    // 不是max的这个是给了padding，可是前向也没给，就反向给了，这么多个kernel就它是这样
    OffsetPreparationFor4Dimension<>(index,
                                     channel_last,
                                     divmods,
                                     padding_width,
                                     padding_height,
                                     output_width,
                                     output_height,
                                     &w_offset,
                                     &h_offset,
                                     &c_offset,
                                     &output_offset);
    if (pool_process.use_x) {
      input = input_data[index];
      output_data += output_offset;
    }
    output_grad += output_offset;

    if (adaptive) {
      // 为什么这里有还要用divmods呢？其他kernel好像不用，包括前向
      auto tmp_phend = divmods.height.Divmod((h_offset + 1) * output_height);
      auto tmp_pwend = divmods.width.Divmod((w_offset + 1) * output_width);
      phstart = divmods.height.Div(h_offset * output_height);
      pwstart = divmods.width.Div(w_offset * output_width);
      // 向上取整
      phend = tmp_phend.val[1] > 0 ? tmp_phend.val[0] + 1 : tmp_phend.val[0];
      pwend = tmp_pwend.val[1] > 0 ? tmp_pwend.val[0] + 1 : tmp_pwend.val[0];

      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          auto ksize_w_divmod = divmods.ksize_w.Divmod(input_width);
          auto ksize_h_divmod = divmods.ksize_h.Divmod(input_height);
          auto tmp_width = ksize_w_divmod.val[1] > 0 ? ksize_w_divmod.val[0] + 1
                                                     : ksize_w_divmod.val[0];
          auto tmp_height = ksize_h_divmod.val[1] > 0
                                ? ksize_h_divmod.val[0] + 1
                                : ksize_h_divmod.val[0];
          int pool_size = tmp_height * tmp_width;
          int tmp_idx = ph * output_width + pw;
          int output_sub_idx =
              channel_last ? tmp_idx * divmods.channel.divisor + c_offset
                           : tmp_idx;
          T ouput_value = pool_process.use_x ? output_data[output_sub_idx]
                                             : static_cast<T>(0);
          pool_process.compute(input,
                               ouput_value,
                               output_grad[output_sub_idx],
                               static_cast<T>(1.0 / pool_size),
                               &input_grad_data);
        }
      }
    } else {
      auto stride_height_div = divmods.stride_h.Div(h_offset - ksize_height);
      auto stride_width_div = divmods.stride_w.Div(w_offset - ksize_width);
      phstart = (h_offset < ksize_height) ? 0 : stride_height_div + 1;
      pwstart = (w_offset < ksize_width) ? 0 : stride_width_div + 1;
      phend = min(divmods.stride_h.Div(h_offset) + 1, output_height);
      pwend = min(divmods.stride_w.Div(w_offset) + 1, output_width);

      if (exclusive) {
        for (int ph = phstart; ph < phend; ++ph) {
          for (int pw = pwstart; pw < pwend; ++pw) {
            int hstart = ph * stride_height - padding_height;
            int wstart = pw * stride_width - padding_width;
            int hend = min(hstart + ksize_height, input_height);
            int wend = min(wstart + ksize_width, input_width);
            hstart = max(hstart, 0);
            wstart = max(wstart, 0);
            int pool_size = (hend - hstart) * (wend - wstart);
            int tmp_idx = ph * output_width + pw;
            int output_sub_idx =
                channel_last ? tmp_idx * divmods.channel.divisor + c_offset
                             : tmp_idx;
            T ouput_value = pool_process.use_x ? output_data[output_sub_idx]
                                               : static_cast<T>(0);
            pool_process.compute(input,
                                 ouput_value,
                                 output_grad[output_sub_idx],
                                 static_cast<T>(1.0 / pool_size),
                                 &input_grad_data);
          }
        }
      } else {
        for (int ph = phstart; ph < phend; ++ph) {
          for (int pw = pwstart; pw < pwend; ++pw) {
            int pool_size = ksize_height * ksize_width;
            int tmp_idx = ph * output_width + pw;
            int output_sub_idx =
                channel_last ? tmp_idx * divmods.channel.divisor + c_offset
                             : tmp_idx;
            T ouput_value = pool_process.use_x ? output_data[output_sub_idx]
                                               : static_cast<T>(0);
            pool_process.compute(input,
                                 ouput_value,
                                 output_grad[output_sub_idx],
                                 static_cast<T>(1.0 / pool_size),
                                 &input_grad_data);
          }
        }
      }
    }
    input_grad[index] = input_grad_data;
  }
}

template <typename T>
__global__ void KernelMaxPool2DGrad(const int nthreads,
                                    const T* input_data,
                                    const T* output_data,
                                    const T* output_grad,
                                    const int channels,
                                    const int input_height,
                                    const int input_width,
                                    const int output_height,
                                    const int output_width,
                                    const int ksize_height,
                                    const int ksize_width,
                                    const int stride_height,
                                    const int stride_width,
                                    const int padding_height,
                                    const int padding_width,
                                    T* input_grad,
                                    FastDivModForPooling divmods,
                                    bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset, h_offset, c_offset, input_offset;
    OffsetPreparationFor4Dimension<FastDivModForPooling>(index,
                                                         channel_last,
                                                         divmods,
                                                         0,
                                                         0,
                                                         input_width,
                                                         input_height,
                                                         &w_offset,
                                                         &h_offset,
                                                         &c_offset,
                                                         &input_offset);
    input_data += input_offset;
    input_grad += input_offset;

    int hstart = h_offset * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = w_offset * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    T ele = output_data[index];
    int maxIndex = -1;
    bool stop = false;
    for (int h = hstart; h < hend && !stop; ++h) {
      for (int w = wstart; w < wend && !stop; ++w) {
        int input_data_idx = channel_last
                                 ? (h * input_width + w) * channels + c_offset
                                 : h * input_width + w;
        if (ele == input_data[input_data_idx]) {
          maxIndex = input_data_idx;
          stop = true;
        }
      }
    }

    if (maxIndex != -1) {
      // atomic add
      paddle::platform::CudaAtomicAdd(input_grad + maxIndex,
                                      output_grad[index]);
    }
  }
}

template <typename PoolProcess, typename T>
void Pool2dDirectCUDAFunctor<PoolProcess, T>::operator()(
    const T* input,
    const std::vector<int>& input_shape,
    const std::vector<int>& output_shape,
    const std::vector<int>& ksize,
    const std::vector<int>& strides,
    const std::vector<int>& paddings,
    bool exclusive,
    bool adaptive,
    T* output,
    gpuStream_t stream,
    PoolProcess pool_compute) {
  const int batch_size = input_shape[0];
  const int input_channels = input_shape[1];
  const int input_height = input_shape[2];
  const int input_width = input_shape[3];
  const int output_channels = output_shape[1];
  const int output_height = output_shape[2];
  const int output_width = output_shape[3];
  const int ksize_height = ksize[0];
  const int ksize_width = ksize[1];
  const int stride_height = strides[0];
  const int stride_width = strides[1];
  const int padding_height = paddings[0];
  const int padding_width = paddings[1];

  int nthreads = batch_size * output_channels * output_height * output_width;
  int thread_num = 1024;
#ifdef WITH_NV_JETSON
  // backends::gpu::ChangeThreadNum(context, &thread_num);
  thread_num = 512;
#endif
  int blocks = (nthreads + thread_num - 1) / thread_num;
  dim3 threads(thread_num, 1);
  dim3 grid(blocks, 1);

  auto pool_divmods =
      FastDivModForPooling(input_channels, output_width, output_height);
  KernelPool2D<PoolProcess, T><<<grid, threads, 0, stream>>>(nthreads,
                                                             input,
                                                             input_channels,
                                                             input_height,
                                                             input_width,
                                                             output_height,
                                                             output_width,
                                                             ksize_height,
                                                             ksize_width,
                                                             stride_height,
                                                             stride_width,
                                                             padding_height,
                                                             padding_width,
                                                             pool_divmods,
                                                             pool_compute,
                                                             exclusive,
                                                             adaptive,
                                                             output);
}

/*
 * Tensors are in NCHW or NHWC format.
 * Ksize, strides are two elements. These two elements represent height
 * and width, respectively.
 * Paddings are four elements. These four elements represent height_up,
 * height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dFunctor<phi::GPUContext, PoolProcess, T> {
 public:
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  bool exclusive,
                  bool adaptive,
                  DenseTensor* output,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = context.template Alloc<T>(output);

    int nthreads = batch_size * output_channels * output_height * output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    backends::gpu::ChangeThreadNum(context, &thread_num);
#endif
    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    auto pool_divmods =
        FastDivModForPooling(input_channels, output_width, output_height);
    KernelPool2D<PoolProcess, T>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 input_data,
                                                 input_channels,
                                                 input_height,
                                                 input_width,
                                                 output_height,
                                                 output_width,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_height,
                                                 stride_width,
                                                 padding_height,
                                                 padding_width,
                                                 pool_divmods,
                                                 pool_process,
                                                 exclusive,
                                                 adaptive,
                                                 output_data);
  }
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format,
                  bool exclusive,
                  bool adaptive,
                  DenseTensor* output,
                  PoolProcess pool_process) {
    bool channel_last = (data_format == "NHWC");
    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[3] : input.dims()[1];
    const int input_height = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_width = channel_last ? input.dims()[2] : input.dims()[3];

    const int output_channels =
        channel_last ? output->dims()[3] : output->dims()[1];
    const int output_height =
        channel_last ? output->dims()[1] : output->dims()[2];
    const int output_width =
        channel_last ? output->dims()[2] : output->dims()[3];

    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];

    const int stride_height = strides[0];
    const int stride_width = strides[1];

    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = context.template Alloc<T>(output);

    int nthreads = batch_size * output_channels * output_height * output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    backends::gpu::ChangeThreadNum(context, &thread_num);
#endif
    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    auto pool_divmods =
        FastDivModForPooling(input_channels, output_width, output_height);
    KernelPool2D<PoolProcess, T>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 input_data,
                                                 input_channels,
                                                 input_height,
                                                 input_width,
                                                 output_height,
                                                 output_width,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_height,
                                                 stride_width,
                                                 padding_height,
                                                 padding_width,
                                                 pool_divmods,
                                                 pool_process,
                                                 exclusive,
                                                 adaptive,
                                                 output_data,
                                                 channel_last);
  }
};
/*
 * Tensors are in NCHW or NHWC format.
 * Ksize, strides are two elements. These two elements represent height
 * and width, respectively.
 * Paddings are four elements. These four elements represent height_up,
 * height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dGradFunctor<phi::GPUContext, PoolProcess, T> {
 public:
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const DenseTensor& output,
                  const DenseTensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  bool exclusive,
                  bool adaptive,
                  DenseTensor* input_grad,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = context.template Alloc<T>(input_grad);

    int nthreads = batch_size * input_channels * input_height * input_width;
    auto pool_divmods = FastDivModForPoolingWithMoreStaff(input_channels,
                                                          input_width,
                                                          input_height,
                                                          ksize_width,
                                                          ksize_height,
                                                          stride_width,
                                                          stride_height);
    
    // SUB:REF:DOING 一维grid和block的起法，只在2d的特定2个kernel使用，可以考虑用到各个kernel
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(context, nthreads);
    KernelPool2DGrad<T, PoolProcess><<<config.block_per_grid,
                                       config.thread_per_block,
                                       0,
                                       context.stream()>>>(nthreads,
                                                           input_data,
                                                           output_data,
                                                           output_grad_data,
                                                           output_width,
                                                           output_height,
                                                           input_width,
                                                           input_height,
                                                           ksize_width,
                                                           ksize_height,
                                                           stride_width,
                                                           stride_height,
                                                           padding_width,
                                                           padding_height,
                                                           pool_divmods,
                                                           pool_process,
                                                           exclusive,
                                                           adaptive,
                                                           input_grad_data);
  }
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const DenseTensor& output,
                  const DenseTensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format,
                  bool exclusive,
                  bool adaptive,
                  DenseTensor* input_grad,
                  PoolProcess pool_process) {
    bool channel_last = (data_format == "NHWC");

    const int batch_size = input.dims()[0];
    const int input_channels = channel_last ? input.dims()[3] : input.dims()[1];
    const int input_height = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_width = channel_last ? input.dims()[2] : input.dims()[3];

    const int output_channels =
        channel_last ? output.dims()[3] : output.dims()[1];
    const int output_height =
        channel_last ? output.dims()[1] : output.dims()[2];
    const int output_width = channel_last ? output.dims()[2] : output.dims()[3];

    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];

    const int stride_height = strides[0];
    const int stride_width = strides[1];

    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = context.template Alloc<T>(input_grad);

    int nthreads = batch_size * input_channels * input_height * input_width;
    auto pool_divmods = FastDivModForPoolingWithMoreStaff(input_channels,
                                                          input_width,
                                                          input_height,
                                                          ksize_width,
                                                          ksize_height,
                                                          stride_width,
                                                          stride_height);

    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(context, nthreads);
    KernelPool2DGrad<T, PoolProcess><<<config.block_per_grid,
                                       config.thread_per_block,
                                       0,
                                       context.stream()>>>(nthreads,
                                                           input_data,
                                                           output_data,
                                                           output_grad_data,
                                                           output_width,
                                                           output_height,
                                                           input_width,
                                                           input_height,
                                                           ksize_width,
                                                           ksize_height,
                                                           stride_width,
                                                           stride_height,
                                                           padding_width,
                                                           padding_height,
                                                           pool_divmods,
                                                           pool_process,
                                                           exclusive,
                                                           adaptive,
                                                           input_grad_data,
                                                           channel_last);
  }
};

/*
 * Tensors are in NCHW or NHWC format.
 * Ksize, strides are two elements. These two elements represent height
 * and width, respectively.
 * Paddings are four elements. These four elements represent height_up,
 * height_down, width_left and width_right, respectively.
 */
template <typename T>
class MaxPool2dGradFunctor<phi::GPUContext, T> {
 public:
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const DenseTensor& output,
                  const DenseTensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  DenseTensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = context.template Alloc<T>(input_grad);

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    auto pool_divmods =
        FastDivModForPooling(input_channels, output_width, output_height);
    KernelMaxPool2DGrad<T>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 input_data,
                                                 output_data,
                                                 output_grad_data,
                                                 input_channels,
                                                 input_height,
                                                 input_width,
                                                 output_height,
                                                 output_width,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_height,
                                                 stride_width,
                                                 padding_height,
                                                 padding_width,
                                                 input_grad_data,
                                                 pool_divmods);
  }
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const DenseTensor& output,
                  const DenseTensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format,
                  DenseTensor* input_grad) {
    bool channel_last = (data_format == "NHWC");

    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[3] : input.dims()[1];
    const int input_height = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_width = channel_last ? input.dims()[2] : input.dims()[3];

    const int output_channels =
        channel_last ? output.dims()[3] : output.dims()[1];
    const int output_height =
        channel_last ? output.dims()[1] : output.dims()[2];
    const int output_width = channel_last ? output.dims()[2] : output.dims()[3];

    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];

    const int stride_height = strides[0];
    const int stride_width = strides[1];

    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = context.template Alloc<T>(input_grad);

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    auto pool_divmods =
        FastDivModForPooling(input_channels, output_width, output_height);

    KernelMaxPool2DGrad<T>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 input_data,
                                                 output_data,
                                                 output_grad_data,
                                                 input_channels,
                                                 input_height,
                                                 input_width,
                                                 output_height,
                                                 output_width,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_height,
                                                 stride_width,
                                                 padding_height,
                                                 padding_width,
                                                 input_grad_data,
                                                 pool_divmods,
                                                 channel_last);
  }
};

template class Pool2dDirectCUDAFunctor<MaxPool<float>, float>;
template class Pool2dDirectCUDAFunctor<AvgPool<float>, float>;

template class MaxPool2dGradFunctor<phi::GPUContext, float>;
template class MaxPool2dGradFunctor<phi::GPUContext, double>;
template class MaxPool2dGradFunctor<phi::GPUContext, dtype::float16>;

template class Pool2dFunctor<phi::GPUContext, MaxPool<float>, float>;
template class Pool2dFunctor<phi::GPUContext, AvgPool<float>, float>;
template class Pool2dGradFunctor<phi::GPUContext, MaxPoolGrad<float>, float>;
template class Pool2dGradFunctor<phi::GPUContext, AvgPoolGrad<float>, float>;
template class Pool2dFunctor<phi::GPUContext, MaxPool<double>, double>;
template class Pool2dFunctor<phi::GPUContext, AvgPool<double>, double>;
template class Pool2dGradFunctor<phi::GPUContext, MaxPoolGrad<double>, double>;
template class Pool2dGradFunctor<phi::GPUContext, AvgPoolGrad<double>, double>;

template class Pool2dFunctor<phi::GPUContext,
                             MaxPool<dtype::float16>,
                             dtype::float16>;
template class Pool2dFunctor<phi::GPUContext,
                             AvgPool<dtype::float16>,
                             dtype::float16>;
template class Pool2dGradFunctor<phi::GPUContext,
                                 MaxPoolGrad<dtype::float16>,
                                 dtype::float16>;
template class Pool2dGradFunctor<phi::GPUContext,
                                 AvgPoolGrad<dtype::float16>,
                                 dtype::float16>;

template <typename PoolProcess, typename T>
__global__ void KernelPool3D(const int nthreads,
                             const T* input_data,
                             const int channels,
                             const int input_depth,
                             const int input_height,
                             const int input_width,
                             const int output_depth,
                             const int output_height,
                             const int output_width,
                             const int ksize_depth,
                             const int ksize_height,
                             const int ksize_width,
                             const int stride_depth,
                             const int stride_height,
                             const int stride_width,
                             const int padding_depth,
                             const int padding_height,
                             const int padding_width,
                             PoolProcess pool_process,
                             bool exclusive,
                             bool adaptive,
                             T* output_data,
                             bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw, ph, pd, c, batch_idx;
    // 这个才是合乎我的逻辑的index方式，直接根据output来计算
    if (!channel_last) {
      pw = index % output_width;
      ph = (index / output_width) % output_height;
      pd = (index / output_width / output_height) % output_depth;
      c = (index / output_width / output_height / output_depth) % channels;
      batch_idx =
          index / output_width / output_height / output_depth / channels;
    } else {
      c = index % channels;
      pw = (index / channels) % output_width;
      ph = (index / channels / output_width) % output_height;
      pd = (index / channels / output_width / output_height) % output_depth;
      batch_idx =
          index / channels / output_width / output_height / output_depth;
    }

    int dstart, dend;
    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      dstart = AdaptStartIndex(pd, input_depth, output_depth);
      dend = AdaptEndIndex(pd, input_depth, output_depth);

      hstart = AdaptStartIndex(ph, input_height, output_height);
      hend = AdaptEndIndex(ph, input_height, output_height);

      wstart = AdaptStartIndex(pw, input_width, output_width);
      wend = AdaptEndIndex(pw, input_width, output_width);
    } else {
      dstart = pd * stride_depth - padding_depth;
      hstart = ph * stride_height - padding_height;
      wstart = pw * stride_width - padding_width;
      dend = min(dstart + ksize_depth, input_depth);
      hend = min(hstart + ksize_height, input_height);
      wend = min(wstart + ksize_width, input_width);
      dstart = max(dstart, 0);
      hstart = max(hstart, 0);
      wstart = max(wstart, 0);
    }

    int input_data_stride;
    if (!channel_last) { /* NCDHW */
      input_data_stride =
          (batch_idx * channels + c) * input_depth * input_height * input_width;
    } else { /* NDHWC */
      input_data_stride =
          batch_idx * input_depth * input_height * input_width * channels;
    }
    input_data += input_data_stride;

    // 就是根据pool的类型（max，avg）创建的一个模版类，用于初始化元素值，对元素值求max/pool
    T ele = pool_process.initial();
    // 每个index对应output一个元素，output一个元素对应在input网格上作用一个kernel
    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          auto input_data_idx =
              channel_last
                  ? ((d * input_height + h) * input_width + w) * channels + c
                  : (d * input_height + h) * input_width + w;
          pool_process.compute(input_data[input_data_idx], &ele);
        }
      }
    }
    int pool_size = (exclusive || adaptive)
                        ? (dend - dstart) * (hend - hstart) * (wend - wstart)
                        : ksize_depth * ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}

template <typename T, typename PoolProcess>
__global__ void KernelPool3DGrad(const int nthreads,
                                 const T* __restrict__ input_data,
                                 const T* __restrict__ output_data,
                                 const T* __restrict__ output_grad,
                                 const int channels,
                                 const int input_depth,
                                 const int input_height,
                                 const int input_width,
                                 const int output_depth,
                                 const int output_height,
                                 const int output_width,
                                 const int ksize_depth,
                                 const int ksize_height,
                                 const int ksize_width,
                                 const int stride_depth,
                                 const int stride_height,
                                 const int stride_width,
                                 const int padding_depth,
                                 const int padding_height,
                                 const int padding_width,
                                 PoolProcess pool_process,
                                 bool exclusive,
                                 bool adaptive,
                                 T* input_grad,
                                 bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset, h_offset, d_offset, c_offset, batch_idx, output_stride;
    T input = static_cast<T>(0);
    if (!channel_last) { /* "NCDHW" */
      w_offset = index % input_width + padding_width;
      h_offset = (index / input_width) % input_height + padding_height;
      d_offset =
          (index / input_width / input_height) % input_depth + padding_depth;
      c_offset = (index / input_width / input_height / input_depth) % channels;
      batch_idx = index / input_width / input_height / input_depth / channels;
      output_stride = (batch_idx * channels + c_offset) * output_depth *
                      output_height * output_width;
    } else { /* "NDHWC" */
      c_offset = index % channels;
      w_offset = (index / channels) % input_width + padding_width;
      h_offset =
          (index / channels / input_width) % input_height + padding_height;
      d_offset = (index / channels / input_width / input_height) % input_depth +
                 padding_depth;
      batch_idx = index / channels / input_width / input_height / input_depth;
      output_stride =
          batch_idx * output_depth * output_height * output_width * channels;
    }

    int pdstart, pdend;
    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      pdstart = AdaptStartIndex(d_offset, output_depth, input_depth);
      pdend = AdaptEndIndex(d_offset, output_depth, input_depth);

      phstart = AdaptStartIndex(h_offset, output_height, input_height);
      phend = AdaptEndIndex(h_offset, output_height, input_height);

      pwstart = AdaptStartIndex(w_offset, output_width, input_width);
      pwend = AdaptEndIndex(w_offset, output_width, input_width);
    } else {
      pdstart = (d_offset < ksize_depth)
                    ? 0
                    : (d_offset - ksize_depth) / stride_depth + 1;
      phstart = (h_offset < ksize_height)
                    ? 0
                    : (h_offset - ksize_height) / stride_height + 1;
      pwstart = (w_offset < ksize_width)
                    ? 0
                    : (w_offset - ksize_width) / stride_width + 1;
      pdend = min((d_offset) / stride_depth + 1, output_depth);
      phend = min((h_offset) / stride_height + 1, output_height);
      pwend = min((w_offset) / stride_width + 1, output_width);
    }
    if (pool_process.use_x) {
      input = input_data[index];
      output_data += output_stride;
    }
    output_grad += output_stride;
    T input_grad_data = static_cast<T>(0.0);

    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          // figure out the pooling size
          int pool_size;
          if (adaptive) {
            pool_size =
                static_cast<int>(
                    ceil(static_cast<double>(input_depth) / ksize_depth)) *
                static_cast<int>(
                    ceil(static_cast<double>(input_height) / ksize_height)) *
                static_cast<int>(
                    ceil(static_cast<double>(input_width) / ksize_width));
          } else {
            int dstart = pd * stride_depth - padding_depth;
            int hstart = ph * stride_height - padding_height;
            int wstart = pw * stride_width - padding_width;
            int dend = min(dstart + ksize_depth, input_depth);
            int hend = min(hstart + ksize_height, input_height);
            int wend = min(wstart + ksize_width, input_width);
            dstart = max(dstart, 0);
            hstart = max(hstart, 0);
            wstart = max(wstart, 0);
            pool_size =
                exclusive ? (dend - dstart) * (hend - hstart) * (wend - wstart)
                          : ksize_depth * ksize_height * ksize_width;
          }

          int output_sub_idx =
              channel_last
                  ? ((pd * output_height + ph) * output_width + pw) * channels +
                        c_offset
                  : (pd * output_height + ph) * output_width + pw;
          T ouput_value = pool_process.use_x ? output_data[output_sub_idx]
                                             : static_cast<T>(0);
          pool_process.compute(input,
                               ouput_value,
                               output_grad[output_sub_idx],
                               static_cast<T>(1.0 / pool_size),
                               &input_grad_data);
        }
      }
    }
    input_grad[index] = input_grad_data;
  }
}

template <typename T>
__global__ void KernelMaxPool3DGrad(const int nthreads,
                                    const T* input_data,
                                    const T* output_data,
                                    const T* output_grad,
                                    const int channels,
                                    const int input_depth,
                                    const int input_height,
                                    const int input_width,
                                    const int output_depth,
                                    const int output_height,
                                    const int output_width,
                                    const int ksize_depth,
                                    const int ksize_height,
                                    const int ksize_width,
                                    const int stride_depth,
                                    const int stride_height,
                                    const int stride_width,
                                    const int padding_depth,
                                    const int padding_height,
                                    const int padding_width,
                                    T* input_grad,
                                    bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw, ph, pd, c, batch_idx;

    if (!channel_last) { /*NCDHW*/
      pw = index % output_width;
      ph = (index / output_width) % output_height;
      pd = (index / output_width / output_height) % output_depth;
      c = (index / output_width / output_height / output_depth) % channels;
      batch_idx =
          index / output_width / output_height / output_depth / channels;
    } else { /*NDHWC*/
      c = index % channels;
      pw = (index / channels) % output_width;
      ph = (index / channels / output_width) % output_height;
      pd = (index / channels / output_width / output_height) % output_depth;
      batch_idx =
          index / channels / output_width / output_height / output_depth;
    }

    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;

    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);

    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);

    T ele = output_data[index];
    bool stop = false;
    int maxIdx = -1;

    int input_stride;
    if (!channel_last) {
      input_stride =
          (batch_idx * channels + c) * input_depth * input_height * input_width;
    } else {
      input_stride =
          batch_idx * input_depth * input_height * input_width * channels;
    }
    input_data += input_stride;
    input_grad += input_stride;
    // SUB:REF:DOING 检查output对应的input数据，如果正好等于当前output数据，说明这个是input的maxindex
    // 为什么不跟pytorch一样直接取mask
    for (int d = dstart; d < dend && !stop; ++d) {
      for (int h = hstart; h < hend && !stop; ++h) {
        for (int w = wstart; w < wend && !stop; ++w) {
          int input_data_idx =
              channel_last
                  ? ((d * input_height + h) * input_width + w) * channels + c
                  : (d * input_height + h) * input_width + w;
          if (ele == input_data[input_data_idx]) {
            stop = true;
            maxIdx = input_data_idx;
          }
        }
      }
    }
    if (maxIdx != -1) {
      // atomic add
      paddle::platform::CudaAtomicAdd(input_grad + maxIdx, output_grad[index]);
    }
  }
}

template <typename PoolProcess, typename T>
void Pool3dDirectCUDAFunctor<PoolProcess, T>::operator()(
    const T* input,
    const std::vector<int>& input_shape,
    const std::vector<int>& output_shape,
    const std::vector<int>& ksize,
    const std::vector<int>& strides,
    const std::vector<int>& paddings,
    bool exclusive,
    bool adaptive,
    T* output,
    gpuStream_t stream,
    PoolProcess pool_compute) {
  const int batch_size = input_shape[0];
  const int input_channels = input_shape[1];
  const int input_depth = input_shape[2];
  const int input_height = input_shape[3];
  const int input_width = input_shape[4];
  const int output_channels = output_shape[1];
  const int output_depth = output_shape[2];
  const int output_height = output_shape[3];
  const int output_width = output_shape[4];
  const int ksize_depth = ksize[0];
  const int ksize_height = ksize[1];
  const int ksize_width = ksize[2];
  const int stride_depth = strides[0];
  const int stride_height = strides[1];
  const int stride_width = strides[2];
  const int padding_depth = paddings[0];
  const int padding_height = paddings[1];
  const int padding_width = paddings[2];

  int nthreads = batch_size * output_channels * output_depth * output_height *
                 output_width;
  int thread_num = 1024;
#ifdef WITH_NV_JETSON
  thread_num = 512;
#endif
  int blocks = (nthreads + thread_num - 1) / thread_num;
  dim3 threads(thread_num, 1);
  dim3 grid(blocks, 1);

  KernelPool3D<PoolProcess, T><<<grid, threads, 0, stream>>>(nthreads,
                                                             input,
                                                             input_channels,
                                                             input_depth,
                                                             input_height,
                                                             input_width,
                                                             output_depth,
                                                             output_height,
                                                             output_width,
                                                             ksize_depth,
                                                             ksize_height,
                                                             ksize_width,
                                                             stride_depth,
                                                             stride_height,
                                                             stride_width,
                                                             padding_depth,
                                                             padding_height,
                                                             padding_width,
                                                             pool_compute,
                                                             exclusive,
                                                             adaptive,
                                                             output);
}

/*
 * Tensors are in NCDHW or NDHWC format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 * Paddings are six elements. These six elements represent depth_forth,
 * depth_back,
 * height_up, height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, class T>
class Pool3dFunctor<phi::GPUContext, PoolProcess, T> {
 public:
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  bool exclusive,
                  bool adaptive,
                  DenseTensor* output,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output->dims()[1];
    const int output_depth = output->dims()[2];
    const int output_height = output->dims()[3];
    const int output_width = output->dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    T* output_data = context.template Alloc<T>(output);

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    backends::gpu::ChangeThreadNum(context, &thread_num);
#endif
    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    KernelPool3D<PoolProcess, T>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 input_data,
                                                 input_channels,
                                                 input_depth,
                                                 input_height,
                                                 input_width,
                                                 output_depth,
                                                 output_height,
                                                 output_width,
                                                 ksize_depth,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_depth,
                                                 stride_height,
                                                 stride_width,
                                                 padding_depth,
                                                 padding_height,
                                                 padding_width,
                                                 pool_process,
                                                 exclusive,
                                                 adaptive,
                                                 output_data);
  }
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format,
                  bool exclusive,
                  bool adaptive,
                  DenseTensor* output,
                  PoolProcess pool_process) {
    bool channel_last = (data_format == "NDHWC");
    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[4] : input.dims()[1];
    const int input_depth = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_height = channel_last ? input.dims()[2] : input.dims()[3];
    const int input_width = channel_last ? input.dims()[3] : input.dims()[4];

    const int output_channels =
        channel_last ? output->dims()[4] : output->dims()[1];
    const int output_depth =
        channel_last ? output->dims()[1] : output->dims()[2];
    const int output_height =
        channel_last ? output->dims()[2] : output->dims()[3];
    const int output_width =
        channel_last ? output->dims()[3] : output->dims()[4];

    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];

    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];

    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    T* output_data = context.template Alloc<T>(output);

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    backends::gpu::ChangeThreadNum(context, &thread_num);
#endif
    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    KernelPool3D<PoolProcess, T>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 input_data,
                                                 input_channels,
                                                 input_depth,
                                                 input_height,
                                                 input_width,
                                                 output_depth,
                                                 output_height,
                                                 output_width,
                                                 ksize_depth,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_depth,
                                                 stride_height,
                                                 stride_width,
                                                 padding_depth,
                                                 padding_height,
                                                 padding_width,
                                                 pool_process,
                                                 exclusive,
                                                 adaptive,
                                                 output_data,
                                                 channel_last);
  }
};

/*
 * Tensors are in NCDHW or NDHWC format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 * Paddings are six elements. These six elements represent depth_forth,
 * depth_back,
 * height_up, height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, class T>
class Pool3dGradFunctor<phi::GPUContext, PoolProcess, T> {
 public:
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const DenseTensor& output,
                  const DenseTensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  bool exclusive,
                  bool adaptive,
                  DenseTensor* input_grad,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = context.template Alloc<T>(input_grad);

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DGrad<T, PoolProcess>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 input_data,
                                                 output_data,
                                                 output_grad_data,
                                                 input_channels,
                                                 input_depth,
                                                 input_height,
                                                 input_width,
                                                 output_depth,
                                                 output_height,
                                                 output_width,
                                                 ksize_depth,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_depth,
                                                 stride_height,
                                                 stride_width,
                                                 padding_depth,
                                                 padding_height,
                                                 padding_width,
                                                 pool_process,
                                                 exclusive,
                                                 adaptive,
                                                 input_grad_data);
  }
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const DenseTensor& output,
                  const DenseTensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format,
                  bool exclusive,
                  bool adaptive,
                  DenseTensor* input_grad,
                  PoolProcess pool_process) {
    bool channel_last = (data_format == "NDHWC");

    const int batch_size = input.dims()[0];
    const int input_channels = channel_last ? input.dims()[4] : input.dims()[1];
    const int input_depth = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_height = channel_last ? input.dims()[2] : input.dims()[3];
    const int input_width = channel_last ? input.dims()[3] : input.dims()[4];

    const int output_channels =
        channel_last ? output.dims()[4] : output.dims()[1];
    const int output_depth = channel_last ? output.dims()[1] : output.dims()[2];
    const int output_height =
        channel_last ? output.dims()[2] : output.dims()[3];
    const int output_width = channel_last ? output.dims()[3] : output.dims()[4];

    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];

    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];

    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = context.template Alloc<T>(input_grad);

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DGrad<T, PoolProcess><<<grid, threads, 0, context.stream()>>>(
        nthreads,
        input_data,
        output_data,
        output_grad_data,
        input_channels,
        input_depth,
        input_height,
        input_width,
        output_depth,
        output_height,
        output_width,
        ksize_depth,
        ksize_height,
        ksize_width,
        stride_depth,
        stride_height,
        stride_width,
        padding_depth,
        padding_height,
        padding_width,
        pool_process,
        exclusive,
        adaptive,
        input_grad_data,
        channel_last);  // add channel_last
  }
};

/*
 * tensors are in NCDHW or NDHWC format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 * Paddings are six elements. These six elements represent depth_forth,
 * depth_back,
 * height_up, height_down, width_left and width_right, respectively.
 */
template <class T>
class MaxPool3dGradFunctor<phi::GPUContext, T> {
 public:
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const DenseTensor& output,
                  const DenseTensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  DenseTensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = context.template Alloc<T>(input_grad);

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DGrad<T>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 input_data,
                                                 output_data,
                                                 output_grad_data,
                                                 input_channels,
                                                 input_depth,
                                                 input_height,
                                                 input_width,
                                                 output_depth,
                                                 output_height,
                                                 output_width,
                                                 ksize_depth,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_depth,
                                                 stride_height,
                                                 stride_width,
                                                 padding_depth,
                                                 padding_height,
                                                 padding_width,
                                                 input_grad_data);
  }
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const DenseTensor& output,
                  const DenseTensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format,
                  DenseTensor* input_grad) {
    bool channel_last = (data_format == "NDHWC");
    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[4] : input.dims()[1];
    const int input_depth = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_height = channel_last ? input.dims()[2] : input.dims()[3];
    const int input_width = channel_last ? input.dims()[3] : input.dims()[4];

    const int output_channels =
        channel_last ? output.dims()[4] : output.dims()[1];
    const int output_depth = channel_last ? output.dims()[1] : output.dims()[2];
    const int output_height =
        channel_last ? output.dims()[2] : output.dims()[3];
    const int output_width = channel_last ? output.dims()[3] : output.dims()[4];

    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];

    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];

    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = context.template Alloc<T>(input_grad);

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads,
        input_data,
        output_data,
        output_grad_data,
        input_channels,
        input_depth,
        input_height,
        input_width,
        output_depth,
        output_height,
        output_width,
        ksize_depth,
        ksize_height,
        ksize_width,
        stride_depth,
        stride_height,
        stride_width,
        padding_depth,
        padding_height,
        padding_width,
        input_grad_data,
        channel_last);  // add channel_last
  }
};

template class Pool3dDirectCUDAFunctor<MaxPool<float>, float>;
template class Pool3dDirectCUDAFunctor<AvgPool<float>, float>;

template class MaxPool3dGradFunctor<phi::GPUContext, float>;
template class MaxPool3dGradFunctor<phi::GPUContext, double>;
template class MaxPool3dGradFunctor<phi::GPUContext, dtype::float16>;

template class Pool3dFunctor<phi::GPUContext, MaxPool<float>, float>;
template class Pool3dFunctor<phi::GPUContext, AvgPool<float>, float>;
template class Pool3dGradFunctor<phi::GPUContext, MaxPoolGrad<float>, float>;
template class Pool3dGradFunctor<phi::GPUContext, AvgPoolGrad<float>, float>;
template class Pool3dFunctor<phi::GPUContext, MaxPool<double>, double>;
template class Pool3dFunctor<phi::GPUContext, AvgPool<double>, double>;
template class Pool3dGradFunctor<phi::GPUContext, MaxPoolGrad<double>, double>;
template class Pool3dGradFunctor<phi::GPUContext, AvgPoolGrad<double>, double>;

template class Pool3dFunctor<phi::GPUContext,
                             MaxPool<dtype::float16>,
                             dtype::float16>;
template class Pool3dFunctor<phi::GPUContext,
                             AvgPool<dtype::float16>,
                             dtype::float16>;
template class Pool3dGradFunctor<phi::GPUContext,
                                 MaxPoolGrad<dtype::float16>,
                                 dtype::float16>;
template class Pool3dGradFunctor<phi::GPUContext,
                                 AvgPoolGrad<dtype::float16>,
                                 dtype::float16>;

// SUB:REF:DOING maxpool2d前向kernel
template <typename T1, typename T2>
__global__ void KernelMaxPool2dWithIdx(const int nthreads,
                                       const T1* input_data,
                                       const int channels,
                                       const int input_height,
                                       const int input_width,
                                       const int output_height,
                                       const int output_width,
                                       const int ksize_height,
                                       const int ksize_width,
                                       const int stride_height,
                                       const int stride_width,
                                       const int padding_height,
                                       const int padding_width,
                                       bool adaptive,
                                       T1* output_data,
                                       T2* mask_data,
                                       FastDivModForPooling divmods) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int hstart, hend, wstart, wend;
    int w_offset, h_offset, c_offset, input_offset;
    // 第二个参数是channel_last，第四个和第五个参数是pad_width和pad_height
    // __device__ void OffsetPreparationFor4Dimension(int index, bool channel_last, FastDivModForPooling divmods, const int pad_width, const int pad_height, const int aux_width, const int aux_height, int* w_offset, int* h_offset, int* c_offset, int* stride)
    // 这里pad给0有点奇怪，不过其实应该也取决于后面怎么算，这里后面又去减pad了
    OffsetPreparationFor4Dimension<FastDivModForPooling>(index,
                                                         false,
                                                         divmods,
                                                         0,
                                                         0,
                                                         input_width,
                                                         input_height,
                                                         &w_offset,
                                                         &h_offset,
                                                         &c_offset,
                                                         &input_offset);
    // SUB:FIXME 这么写不对吧，如果多次循环的话，index每次更新以后算出来的并不是stride，而是相对传参进来的位置的offset
    // 原本写的kernel的一维线程配置刚好也是计算好只循环1次的
    input_data += input_offset;

    // 这里前向是用了adaptive的分装好的index计算接口
    if (adaptive) {
      hstart = AdaptStartIndex(h_offset, input_height, output_height);
      hend = AdaptEndIndex(h_offset, input_height, output_height);

      wstart = AdaptStartIndex(w_offset, input_width, output_width);
      wend = AdaptEndIndex(w_offset, input_width, output_width);
    } else {
      // offset是相对output算的，对于input，kernel作用范围要往回走padding
      hstart = h_offset * stride_height - padding_height;
      hend = min(hstart + ksize_height, input_height);
      hstart = max(hstart, 0);

      wstart = w_offset * stride_width - padding_width;
      wend = min(wstart + ksize_width, input_width);
      wstart = max(wstart, 0);
    }

    T1 ele = -FLT_MAX;
    int max_index = -1;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int input_index = h * input_width + w;
        if (ele < input_data[input_index]) {
          max_index = input_index;
          ele = input_data[input_index];
        }
      }
    }
    output_data[index] = ele;
    mask_data[index] = max_index;
  }
}

// SUB:REF:DOING maxpool2d反向kernel
template <typename T1, typename T2>
__global__ void KernelMaxPool2DWithIdxGrad(const int nthreads,
                                           const T1* output_grad,
                                           const T2* mask_data,
                                           const int channels,
                                           const int input_height,
                                           const int input_width,
                                           const int output_height,
                                           const int output_width,
                                           const int ksize_height,
                                           const int ksize_width,
                                           const int stride_height,
                                           const int stride_width,
                                           const int padding_height,
                                           const int padding_width,
                                           bool adaptive,
                                           T1* input_grad,
                                           FastDivModForPooling divmods) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int phstart, phend, pwstart, pwend;
    int w_offset, h_offset, c_offset, output_offset;
    // 跟前向一样也没有给pad
    OffsetPreparationFor4Dimension<FastDivModForPooling>(index,
                                                         false,
                                                         divmods,
                                                         0,
                                                         0,
                                                         output_width,
                                                         output_height,
                                                         &w_offset,
                                                         &h_offset,
                                                         &c_offset,
                                                         &output_offset);
    // SUB:FIXME 这么写不对吧，如果多次循环的话，index每次更新以后算出来的并不是stride，而是相对传参进来的位置的offset
    mask_data += output_offset;
    output_grad += output_offset;

    // 反向没有用adaptive索引计算的封装接口
    if (adaptive) {
      phstart = h_offset * output_height / input_height;
      phend =
          min((h_offset + 1) * output_height / input_height + 1, output_height);
      pwstart = w_offset * output_width / input_width;
      pwend =
          min((w_offset + 1) * output_width / input_width + 1, output_width);
    } else {
      // 这里是要算output即grad_input的索引，所以要加padding
      // 注意output_grad指的就是output的梯度
      phstart =
          (h_offset + padding_height < ksize_height)
              ? 0
              : (h_offset + padding_height - ksize_height) / stride_height + 1;
      pwstart =
          (w_offset + padding_width < ksize_width)
              ? 0
              : (w_offset + padding_width - ksize_width) / stride_width + 1;
      phend =
          min((h_offset + padding_height) / stride_height + 1, output_height);
      pwend = min((w_offset + padding_width) / stride_width + 1, output_width);
    }

    T1 input_grad_data = 0;
    int input_current_featuremap_idx = h_offset * input_width + w_offset;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (mask_data[ph * output_width + pw] == input_current_featuremap_idx)
          input_grad_data += output_grad[ph * output_width + pw];
      }
    }
    // 根据input切的grid和block
    // input的这个位置有没有梯度，其实也是取决于output上所有它可能作用到的位置是否是max，求和
    input_grad[index] = input_grad_data;
  }
}

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
 // SUB:REF:DOING maxpool2d起前向
template <typename T1, typename T2>
class MaxPool2dWithIndexFunctor<phi::GPUContext, T1, T2> {
 public:
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  bool adaptive,
                  DenseTensor* output,
                  DenseTensor* mask) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T1* input_data = input.data<T1>();
    T1* output_data = context.template Alloc<T1>(output);
    T2* mask_data = context.template Alloc<T2>(mask);

    int nthreads = batch_size * output_channels * output_height * output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    backends::gpu::ChangeThreadNum(context, &thread_num);
#endif

    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    // 为什么只需要这三个参数呢。。如果input_channels是output_channels的话倒还能理解
    // 不过注意看output_channels都没有传到前向kernel里，对于pooling操作，input_channels==output_channels吧，没有index的2d前向同样也是这么写的
    auto pool_divmods =
        FastDivModForPooling(input_channels, output_width, output_height);
    // 这里要把pool_divmods作为参数给进去
    KernelMaxPool2dWithIdx<T1, T2>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 input_data,
                                                 input_channels,
                                                 input_height,
                                                 input_width,
                                                 output_height,
                                                 output_width,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_height,
                                                 stride_width,
                                                 padding_height,
                                                 padding_width,
                                                 adaptive,
                                                 output_data,
                                                 mask_data,
                                                 pool_divmods);
  }
};

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
 // SUB:REF:DOING maxpool2d起反向
template <typename T1, typename T2>
class MaxPool2dWithIndexGradFunctor<phi::GPUContext, T1, T2> {
 public:
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& output_grad,
                  const DenseTensor& mask,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  bool adaptive,
                  DenseTensor* input_grad) {
    const int batch_size = input_grad->dims()[0];
    const int input_channels = input_grad->dims()[1];
    const int input_height = input_grad->dims()[2];
    const int input_width = input_grad->dims()[3];
    const int output_height = output_grad.dims()[2];
    const int output_width = output_grad.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T2* mask_data = mask.data<T2>();
    const T1* output_grad_data = output_grad.data<T1>();
    T1* input_grad_data = context.template Alloc<T1>(input_grad);

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    // 跟前向逻辑是一样的，就是把计算索引需要的output传进去，对于反向而言input就是output
    auto pool_divmods =
        FastDivModForPooling(input_channels, input_width, input_height);
    KernelMaxPool2DWithIdxGrad<T1, T2>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 output_grad_data,
                                                 mask_data,
                                                 input_channels,
                                                 input_height,
                                                 input_width,
                                                 output_height,
                                                 output_width,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_height,
                                                 stride_width,
                                                 padding_height,
                                                 padding_width,
                                                 adaptive,
                                                 input_grad_data,
                                                 pool_divmods);
  }
};

template class MaxPool2dWithIndexFunctor<phi::GPUContext, float, int>;
template class MaxPool2dWithIndexGradFunctor<phi::GPUContext, float, int>;
template class MaxPool2dWithIndexFunctor<phi::GPUContext, double, int>;
template class MaxPool2dWithIndexGradFunctor<phi::GPUContext, double, int>;

// SUB:DONE maxpool3d前向kernel
/*
template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdx(const int nthreads,
                                       const T1* input_data,
                                       const int channels,
                                       const int input_depth,
                                       const int input_height,
                                       const int input_width,
                                       const int output_depth,
                                       const int output_height,
                                       const int output_width,
                                       const int ksize_depth,
                                       const int ksize_height,
                                       const int ksize_width,
                                       const int stride_depth,
                                       const int stride_height,
                                       const int stride_width,
                                       const int padding_depth,
                                       const int padding_height,
                                       const int padding_width,
                                       bool adaptive,
                                       T1* output_data,
                                       T2* mask_data,
                                       FastDivModForPooling3D divmods) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int dstart, dend, hstart, hend, wstart, wend;
    int w_offset, h_offset, d_offset, c_offset, input_offset;
    OffsetPreparationFor5Dimension<FastDivModForPooling3D>(index, 
                                                           false, 
                                                           divmods, 
                                                           0, 
                                                           0,
                                                           0,
                                                           input_width, 
                                                           input_height, 
                                                           input_depth, 
                                                           &w_offset, 
                                                           &h_offset, 
                                                           &d_offset, 
                                                           &c_offset, 
                                                           &input_offset);
    input_data += input_offset;

    if (adaptive) {
      dstart = AdaptStartIndex(d_offset, input_depth, output_depth);
      dend = AdaptEndIndex(d_offset, input_depth, output_depth);

      hstart = AdaptStartIndex(h_offset, input_height, output_height);
      hend = AdaptEndIndex(h_offset, input_height, output_height);

      wstart = AdaptStartIndex(w_offset, input_width, output_width);
      wend = AdaptEndIndex(w_offset, input_width, output_width);
    } else {
      dstart = d_offset * stride_depth - padding_depth;
      hstart = h_offset * stride_height - padding_height;
      wstart = w_offset * stride_width - padding_width;
      dend = min(dstart + ksize_depth, input_depth);
      hend = min(hstart + ksize_height, input_height);
      wend = min(wstart + ksize_width, input_width);
      dstart = max(dstart, 0);
      hstart = max(hstart, 0);
      wstart = max(wstart, 0);
    }

    T1 ele = -FLT_MAX;
    int max_index = -1;
    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          // 跟return_mask==false的正向的区别也就在于，把max的逻辑放进来而不是在pool_process里，从而得到index
          if (ele < input_data[(d * input_height + h) * input_width + w]) {
            max_index = (d * input_height + h) * input_width + w;
            ele = input_data[max_index];
          }
        }
      }
    }
    output_data[index] = ele;
    mask_data[index] = max_index;
  }
}
*/

// SUB:DOING 三维线程配置版的maxpool3d前向kernel
template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdx(const int ncd,
                                       const T1* input_data,
                                       const int channels,
                                       const int input_depth,
                                       const int input_height,
                                       const int input_width,
                                       const int output_depth,
                                       const int output_height,
                                       const int output_width,
                                       const int ksize_depth,
                                       const int ksize_height,
                                       const int ksize_width,
                                       const int stride_depth,
                                       const int stride_height,
                                       const int stride_width,
                                       const int padding_depth,
                                       const int padding_height,
                                       const int padding_width,
                                       bool adaptive,
                                       T1* output_data,
                                       T2* mask_data,
                                       FastDivModForPooling3D divmods_output) {
  int w_offset, h_offset, d_offset, nc_offset;
  int dstart, dend, hstart, hend, wstart, wend;
  const T1* input_data_cur;

  w_offset = blockIdx.x * blockDim.x + threadIdx.x;
  h_offset = blockIdx.y * blockDim.y + threadIdx.y;

  if (w_offset < output_width && h_offset < output_height) {
    for (int index_z = blockIdx.z * blockDim.z + threadIdx.z; index_z < ncd; index_z += gridDim.z * blockDim.z) {
      auto output_depth_divmod = divmods_output.depth.Divmod(index_z);
      d_offset = output_depth_divmod.val[1];
      nc_offset = output_depth_divmod.val[0];
      int output_index = nc_offset * output_depth * output_height * output_width + d_offset * output_height * output_width + h_offset * output_width + w_offset;
      int input_offset = nc_offset * input_depth * input_height * input_width;
      input_data_cur = input_data + input_offset;

      if (adaptive) {
        dstart = AdaptStartIndex(d_offset, input_depth, output_depth);
        dend = AdaptEndIndex(d_offset, input_depth, output_depth);
  
        hstart = AdaptStartIndex(h_offset, input_height, output_height);
        hend = AdaptEndIndex(h_offset, input_height, output_height);
  
        wstart = AdaptStartIndex(w_offset, input_width, output_width);
        wend = AdaptEndIndex(w_offset, input_width, output_width);
      } else {
        dstart = d_offset * stride_depth - padding_depth;
        hstart = h_offset * stride_height - padding_height;
        wstart = w_offset * stride_width - padding_width;
        dend = min(dstart + ksize_depth, input_depth);
        hend = min(hstart + ksize_height, input_height);
        wend = min(wstart + ksize_width, input_width);
        dstart = max(dstart, 0);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
      }

      T1 ele = -FLT_MAX;
      int max_index = -1;
      for (int d = dstart; d < dend; ++d) {
        for (int h = hstart; h < hend; ++h) {
          for (int w = wstart; w < wend; ++w) {
            if (ele < input_data_cur[(d * input_height + h) * input_width + w]) {
              max_index = (d * input_height + h) * input_width + w;
              ele = input_data_cur[max_index];
            }
          }
        }
      }
      output_data[output_index] = ele;
      mask_data[output_index] = max_index;
    }
  }
}

// SUB:DONE maxpool3d反向kernel
/*
template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdxGrad(const int nthreads,
                                           const T1* output_grad,
                                           const T2* mask,
                                           const int channels,
                                           const int input_depth,
                                           const int input_height,
                                           const int input_width,
                                           const int output_depth,
                                           const int output_height,
                                           const int output_width,
                                           const int ksize_depth,
                                           const int ksize_height,
                                           const int ksize_width,
                                           const int stride_depth,
                                           const int stride_height,
                                           const int stride_width,
                                           const int padding_depth,
                                           const int padding_height,
                                           const int padding_width,
                                           bool adaptive,
                                           T1* input_grad,
                                           FastDivModForPooling3D divmods,
                                           FastDivModForPooling3DStride divmods_stride) {
  // input参数和stride参数其实不传都可以，因为通过divmods已经传进来了
  // 注意这个是反向，maxpooling的反向是，如果input正好等于output的值，在这个input位置上的梯度就是保持不变，其他的都是0
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    // ncdhw
    // 对于反向而言，input是output，output是input
    // 不知道为啥没有跟其他接口一样考虑ndhwc格式的索引计算？with_index的都没考虑
    // 优化索引计算中的除法和求模，通过fastdivmod计算（前面的函数参数和后面的变量名也需要修改）

    // 注意思考这里的索引计算方式，它的offset和p实际上针对的是grad_input即output来算的
    // 首先这里的grid和block是按照input来切分的，output的多个位置都可能对应到input的某个位置，所以就是循环这些位置，求和来对应到那个input位置的值
    // offset其实也是input的offset

    int pdstart, pdend, phstart, phend, pwstart, pwend;
    int w_offset, h_offset, d_offset, c_offset, output_offset;
    // 注意这里传到output索引并不是说求模的是output哦，实际上求模的是创建divmods是的input索引，output索引是用于计算output_offset
    OffsetPreparationFor5Dimension<FastDivModForPooling3D>(index, 
                                                           false, 
                                                           divmods, 
                                                           0, 
                                                           0,
                                                           0,
                                                           output_width, 
                                                           output_height, 
                                                           output_depth, 
                                                           &w_offset, 
                                                           &h_offset, 
                                                           &d_offset, 
                                                           &c_offset, 
                                                           &output_offset);
    // 后面是索引dhw，所以先把nc给偏移出去
    mask += output_offset;
    output_grad += output_offset;

    // 调用的时候是关掉adapative的，但adaptive是什么？意思就是不需要根据padding，stride，直接就根据input和output算出来kernel的作用范围
    // 这里不知道为什么没有像其他kernel一样直接用封装好的adaptive_index计算接口，不过这个应该也没什么好优化的
    // SUB:TODO adaptive部分也可以用fastdivmod优化下
    if (adaptive) {
      pdstart = d_offset * output_depth / input_depth;
      pdend =
          min((d_offset + 1) * output_depth / input_depth + 1, output_depth);
      phstart = h_offset * output_height / input_height;
      phend =
          min((h_offset + 1) * output_height / input_height + 1, output_height);
      pwstart = w_offset * output_width / input_width;
      pwend =
          min((w_offset + 1) * output_width / input_width + 1, output_width);
    } else {
      // 计算offset的时候，其实锚定原始的input网格，现在是根据对input的padding和kernel_size和stride，重新锚定到output的网格，从移动的视角
      // 用fastdivmod优化这一部分除法，三目运算符就是语法糖，并不是说有更好的性能，可以放心优化
      if (d_offset + padding_depth < ksize_depth) 
        pdstart = 0;
      else 
        pdstart = divmods_stride.depth.Div(d_offset + padding_depth - ksize_depth) + 1;
      
      if (h_offset + padding_height < ksize_height) 
        phstart = 0;
      else
        phstart = divmods_stride.height.Div(h_offset + padding_height - ksize_height) + 1;
  
      if (w_offset + padding_width < ksize_width)
        pwstart = 0;
      else
        pwstart = divmods_stride.width.Div(w_offset + padding_width - ksize_width) + 1;
      
      pdend = min(divmods_stride.depth.Div(d_offset + padding_depth) + 1, output_depth);
      phend = min(divmods_stride.height.Div(h_offset + padding_height) + 1, output_height);
      pwend = min(divmods_stride.width.Div(w_offset + padding_width) + 1, output_width);
    }

    T1 input_grad_data = 0;
    int input_current_feature_map_idx =
        (d_offset * input_height + h_offset) * input_width + w_offset;
    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          // mask存的是跟output大小一样的，对应到input的最大索引
          if (mask[(pd * output_height + ph) * output_width + pw] ==
              input_current_feature_map_idx)
            input_grad_data +=
                output_grad[(pd * output_height + ph) * output_width + pw];
        }
      }
    }
    input_grad[index] = input_grad_data;
  }
}
*/

// SUB:DONE 三维线程配置版的maxpool3d反向kernel
/*
template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdxGrad(const int ncd,
                                           const T1* output_grad,
                                           const T2* mask,
                                           const int channels,
                                           const int input_depth,
                                           const int input_height,
                                           const int input_width,
                                           const int output_depth,
                                           const int output_height,
                                           const int output_width,
                                           const int ksize_depth,
                                           const int ksize_height,
                                           const int ksize_width,
                                           const int stride_depth,
                                           const int stride_height,
                                           const int stride_width,
                                           const int padding_depth,
                                           const int padding_height,
                                           const int padding_width,
                                           bool adaptive,
                                           T1* input_grad,
                                           FastDivModForPooling3D divmods_input,
                                           FastDivModForPooling3DStride divmods_stride) {
  // input参数和stride参数其实不传都可以，因为通过divmods已经传进来了
  // 注意这个是反向，maxpooling的反向是，如果input正好等于output的值，在这个input位置上的梯度就是保持不变，其他的都是0

  // 注意思考这里的索引计算方式，它的offset和p实际上针对的是grad_input即output来算的
  // 首先这里的grid和block是按照input来切分的，output的多个位置都可能对应到input的某个位置，所以就是循环这些位置，求和来对应到那个input位置的值
  // offset其实也是input的offset

  int w_offset, h_offset, d_offset, c_offset, output_offset; 
  int pdstart, pdend, phstart, phend, pwstart, pwend;

  w_offset = blockIdx.x * blockDim.x + threadIdx.x;
  h_offset = blockIdx.y * blockDim.y + threadIdx.y;

  // 一直没有注意到这一点，这里要注意线程配置的时候会向2次幂取整，但实际index的时候不能越界
  if (w_offset < input_width && h_offset < input_height) {
    // SUB:TODO 这里还可以减少除法和求模次数
    // 这样多次循环会不会造成除法次数太多呢？我可以再抽出一个stride计算，每次offset+stride就好，但只循环一次的情况反而更慢
    for (int index_z = blockIdx.z * blockDim.z + threadIdx.z; index_z < ncd; index_z += gridDim.z * blockDim.z) {
        auto input_depth_divmod = divmods_input.depth.Divmod(index_z);
        auto channel_divmod = divmods_input.channel.Divmod(input_depth_divmod.val[0]);
        // SUB:TODO 这里其实不需要计算c_offset，直接nc_offset就好
        d_offset = input_depth_divmod.val[1];
        c_offset = channel_divmod.val[1];
        output_offset = (channel_divmod.val[0] * divmods_input.channel.divisor + c_offset) * output_depth * output_height * output_width;

        if (adaptive) {
          pdstart = divmods_input.depth.Div(d_offset * output_depth);
          pdend = min(divmods_input.depth.Div((d_offset + 1) * output_depth) + 1, output_depth);
          phstart = divmods_input.height.Div(h_offset * output_height);
          phend = min(divmods_input.height.Div((h_offset + 1) * output_height) + 1, output_height);
          pwstart = divmods_input.width.Div(w_offset * output_width);
          pwend = min(divmods_input.width.Div((w_offset + 1) * output_width) + 1, output_width);
        } else {
          // 计算offset的时候，其实锚定原始的input网格，现在是根据对input的padding和kernel_size和stride，重新锚定到output的网格，从移动的视角
          pdstart = (d_offset + padding_depth < ksize_depth) ? 0 : divmods_stride.depth.Div(d_offset + padding_depth - ksize_depth) + 1;
          phstart = (h_offset + padding_height < ksize_height) ? 0 : divmods_stride.height.Div(h_offset + padding_height - ksize_height) + 1;
          pwstart = (w_offset + padding_width < ksize_width) ? 0 : divmods_stride.width.Div(w_offset + padding_width - ksize_width) + 1;
          pdend = min(divmods_stride.depth.Div(d_offset + padding_depth) + 1, output_depth);
          phend = min(divmods_stride.height.Div(h_offset + padding_height) + 1, output_height);
          pwend = min(divmods_stride.width.Div(w_offset + padding_width) + 1, output_width);
        }

        T1 input_grad_data = 0;
        int input_current_feature_map_idx =
            (d_offset * input_height + h_offset) * input_width + w_offset;
        for (int pd = pdstart; pd < pdend; ++pd) {
          for (int ph = phstart; ph < phend; ++ph) {
            for (int pw = pwstart; pw < pwend; ++pw) {
              // mask存的是跟output大小一样的，对应到input的最大索引
              // 这里对output_offset的处理也是，实际上是相对最开始位置的offset，而不是每次循环的stride，所以不能用+=
              if (mask[output_offset + (pd * output_height + ph) * output_width + pw] ==
                  input_current_feature_map_idx)
                input_grad_data +=
                    output_grad[output_offset + (pd * output_height + ph) * output_width + pw];
            }
          }
        }
        input_grad[(index_z * input_height + h_offset) * input_width + w_offset] = input_grad_data;
    }
  }
}
*/

// SUB:DONE pytorch版的maxpool3d反向kernel
template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdxGrad(const int ncd,
                                           const T1* output_grad,
                                           const T2* mask,
                                           const int channels,
                                           const int input_depth,
                                           const int input_height,
                                           const int input_width,
                                           const int output_depth,
                                           const int output_height,
                                           const int output_width,
                                           const int ksize_depth,
                                           const int ksize_height,
                                           const int ksize_width,
                                           const int stride_depth,
                                           const int stride_height,
                                           const int stride_width,
                                           const int padding_depth,
                                           const int padding_height,
                                           const int padding_width,
                                           bool adaptive,
                                           T1* input_grad,
                                           FastDivModForPooling3D divmods_output) {
  int w_offset, h_offset, d_offset, nc_offset; 

  w_offset = blockIdx.x * blockDim.x + threadIdx.x;
  h_offset = blockIdx.y * blockDim.y + threadIdx.y;

  // 一直没有注意到这一点，这里要注意线程配置的时候会向2次幂取整，但实际index的时候不能越界
  if (w_offset < output_width && h_offset < output_height) {
    for (int index_z = blockIdx.z * blockDim.z + threadIdx.z; index_z < ncd; index_z += gridDim.z * blockDim.z) {
      auto output_depth_divmod = divmods_output.depth.Divmod(index_z);
      d_offset = output_depth_divmod.val[1];
      nc_offset = output_depth_divmod.val[0];
      int output_index = nc_offset * output_depth * output_height * output_width + d_offset * output_height * output_width + h_offset * output_width + w_offset;
      int max_index = mask[output_index];
      if (max_index != -1) {
        paddle::platform::CudaAtomicAdd(&input_grad[nc_offset * input_depth * input_height * input_width + max_index], output_grad[output_index]);
      }
    }
  }
}

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
 // SUB:DONE maxpool3d起前向
 /*
template <typename T1, typename T2>
class MaxPool3dWithIndexFunctor<phi::GPUContext, T1, T2> {
 public:
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& input,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  bool adaptive,
                  DenseTensor* output,
                  DenseTensor* mask) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output->dims()[1];
    const int output_depth = output->dims()[2];
    const int output_height = output->dims()[3];
    const int output_width = output->dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T1* input_data = input.data<T1>();
    // 这是什么语法，没看懂
    T1* output_data = context.template Alloc<T1>(output);
    T2* mask_data = context.template Alloc<T2>(mask);

    // 按照output元素的数量设置的线程数
    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    backends::gpu::ChangeThreadNum(context, &thread_num);
#endif

    // 理论上需要更通用的计算方式，但考虑到测例不多，而且每个block尽可能多线程可以减少切换开销，所以这里的优化空间可能并不大
    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    // pool_divmods传到前向kernel
    auto pool_divmods = FastDivModForPooling3D(input_channels, output_width, output_height, output_depth);

    KernelMaxPool3DWithIdx<T1, T2>
        <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                 input_data,
                                                 input_channels,
                                                 input_depth,
                                                 input_height,
                                                 input_width,
                                                 output_depth,
                                                 output_height,
                                                 output_width,
                                                 ksize_depth,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_depth,
                                                 stride_height,
                                                 stride_width,
                                                 padding_depth,
                                                 padding_height,
                                                 padding_width,
                                                 adaptive,
                                                 output_data,
                                                 mask_data,
                                                 pool_divmods);
  }
};
*/

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
 // SUB:DOING 三维线程配置版的maxpool3d起前向
 template <typename T1, typename T2>
 class MaxPool3dWithIndexFunctor<phi::GPUContext, T1, T2> {
  public:
   void operator()(const phi::GPUContext& context,
                   const DenseTensor& input,
                   const std::vector<int>& ksize,
                   const std::vector<int>& strides,
                   const std::vector<int>& paddings,
                   bool adaptive,
                   DenseTensor* output,
                   DenseTensor* mask) {
     const int batch_size = input.dims()[0];
     const int input_channels = input.dims()[1];
     const int input_depth = input.dims()[2];
     const int input_height = input.dims()[3];
     const int input_width = input.dims()[4];
     const int output_channels = output->dims()[1];
     const int output_depth = output->dims()[2];
     const int output_height = output->dims()[3];
     const int output_width = output->dims()[4];
     const int ksize_depth = ksize[0];
     const int ksize_height = ksize[1];
     const int ksize_width = ksize[2];
     const int stride_depth = strides[0];
     const int stride_height = strides[1];
     const int stride_width = strides[2];
     const int padding_depth = paddings[0];
     const int padding_height = paddings[1];
     const int padding_width = paddings[2];
 
     const T1* input_data = input.data<T1>();
     T1* output_data = context.template Alloc<T1>(output);
     T2* mask_data = context.template Alloc<T2>(mask);
 
     int ncd = batch_size * input_channels * output_depth;
 
    //  backends::gpu::GpuLaunchConfig config = backends::gpu::GetGpuLaunchConfig3D(context, ncd, output_height, output_width);
    //  dim3 threads = config.thread_per_block;
    //  dim3 grid = config.block_per_grid;

     int thread_x = 32;
     int thread_y = 8;
     int thread_z = 1;
     dim3 threads(thread_x, thread_y, thread_z);
     std::array<int, 3> max_grid_dim = context.GetCUDAMaxGridDimSize();
     int block_x = (output_width + threads.x - 1) / threads.x;
     int block_y = (output_height + threads.y - 1) / threads.y;
     int block_z = (ncd > max_grid_dim[2] * threads.z) ? max_grid_dim[2] : (ncd + threads.z - 1) / threads.z;
     dim3 grid(block_x, block_y, block_z);
 
     auto pool_divmods_output = FastDivModForPooling3D(input_channels, output_width, output_height, output_depth);
 
     KernelMaxPool3DWithIdx<T1, T2>
         <<<grid, threads, 0, context.stream()>>>(ncd,
                                                  input_data,
                                                  input_channels,
                                                  input_depth,
                                                  input_height,
                                                  input_width,
                                                  output_depth,
                                                  output_height,
                                                  output_width,
                                                  ksize_depth,
                                                  ksize_height,
                                                  ksize_width,
                                                  stride_depth,
                                                  stride_height,
                                                  stride_width,
                                                  padding_depth,
                                                  padding_height,
                                                  padding_width,
                                                  adaptive,
                                                  output_data,
                                                  mask_data,
                                                  pool_divmods_output);
   }
 };

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
 // SUB:DONE maxpool3d起反向
 /*
 template <typename T1, typename T2>
 class MaxPool3dWithIndexGradFunctor<phi::GPUContext, T1, T2> {
  public:
   void operator()(const phi::GPUContext& context,
                   const DenseTensor& output_grad,
                   const DenseTensor& mask,
                   const std::vector<int>& ksize,
                   const std::vector<int>& strides,
                   const std::vector<int>& paddings,
                   bool adaptive,
                   DenseTensor* input_grad) {
     const int batch_size = input_grad->dims()[0];
     const int input_channels = input_grad->dims()[1];
     const int input_depth = input_grad->dims()[2];
     const int input_height = input_grad->dims()[3];
     const int input_width = input_grad->dims()[4];
     const int output_depth = output_grad.dims()[2];
     const int output_height = output_grad.dims()[3];
     const int output_width = output_grad.dims()[4];
     const int ksize_depth = ksize[0];
     const int ksize_height = ksize[1];
     const int ksize_width = ksize[2];
     const int stride_depth = strides[0];
     const int stride_height = strides[1];
     const int stride_width = strides[2];
     const int padding_depth = paddings[0];
     const int padding_height = paddings[1];
     const int padding_width = paddings[2];
 
     const T1* output_grad_data = output_grad.data<T1>();
     const T2* mask_data = mask.data<T2>();
     // 怪怪的，看起来是alloc，但不是应该copy吗
     T1* input_grad_data = context.template Alloc<T1>(input_grad);

     int nthreads =
         batch_size * input_channels * input_depth * input_height * input_width;
     int blocks = (nthreads + 1024 - 1) / 1024;
     dim3 threads(1024, 1);
     dim3 grid(blocks, 1);

     // pool_divmods传到反向kernel
     auto pool_divmods = FastDivModForPooling3D(input_channels, input_width, input_height, input_depth);

     // pool_stride_divmods传到反向kernel
     auto pool_stride_divmods = FastDivModForPooling3DStride(stride_width, stride_height, stride_depth);
 
     KernelMaxPool3DWithIdxGrad<T1, T2>
         <<<grid, threads, 0, context.stream()>>>(nthreads,
                                                  output_grad_data,
                                                  mask_data,
                                                  input_channels,
                                                  input_depth,
                                                  input_height,
                                                  input_width,
                                                  output_depth,
                                                  output_height,
                                                  output_width,
                                                  ksize_depth,
                                                  ksize_height,
                                                  ksize_width,
                                                  stride_depth,
                                                  stride_height,
                                                  stride_width,
                                                  padding_depth,
                                                  padding_height,
                                                  padding_width,
                                                  adaptive,
                                                  input_grad_data,
                                                  pool_divmods,
                                                  pool_stride_divmods);
   }
 };
 */

 /*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
 // SUB:DONE 三维线程配置版的maxpool3d起反向
 /*
template <typename T1, typename T2>
class MaxPool3dWithIndexGradFunctor<phi::GPUContext, T1, T2> {
 public:
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& output_grad,
                  const DenseTensor& mask,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  bool adaptive,
                  DenseTensor* input_grad) {
    const int batch_size = input_grad->dims()[0];
    const int input_channels = input_grad->dims()[1];
    const int input_depth = input_grad->dims()[2];
    const int input_height = input_grad->dims()[3];
    const int input_width = input_grad->dims()[4];
    const int output_depth = output_grad.dims()[2];
    const int output_height = output_grad.dims()[3];
    const int output_width = output_grad.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T1* output_grad_data = output_grad.data<T1>();
    const T2* mask_data = mask.data<T2>();
    T1* input_grad_data = context.template Alloc<T1>(input_grad);

    int ncd = batch_size * input_channels * input_depth;

    backends::gpu::GpuLaunchConfig config = backends::gpu::GetGpuLaunchConfig3D(context, ncd, input_height, input_width);
    dim3 threads = config.thread_per_block;
    dim3 grid = config.block_per_grid;

    // int thread_x = 32;
    // int thread_y = 8;
    // int thread_z = 1;
    // dim3 threads(thread_x, thread_y, thread_z);
    // std::array<int, 3> max_grid_dim = context.GetCUDAMaxGridDimSize();
    // int block_x = (input_width + threads.x - 1) / threads.x;
    // int block_y = (input_height + threads.y - 1) / threads.y;
    // int block_z = (ncd > max_grid_dim[2] * threads.z) ? max_grid_dim[2] : (ncd + threads.z - 1) / threads.z;
    // dim3 grid(block_x, block_y, block_z);
      
    auto pool_divmods_input = FastDivModForPooling3D(input_channels, input_width, input_height, input_depth);
    auto pool_divmods_stride = FastDivModForPooling3DStride(stride_width, stride_height, stride_depth);

    KernelMaxPool3DWithIdxGrad<T1, T2>
        <<<grid, threads, 0, context.stream()>>>(ncd,
                                                 output_grad_data,
                                                 mask_data,
                                                 input_channels,
                                                 input_depth,
                                                 input_height,
                                                 input_width,
                                                 output_depth,
                                                 output_height,
                                                 output_width,
                                                 ksize_depth,
                                                 ksize_height,
                                                 ksize_width,
                                                 stride_depth,
                                                 stride_height,
                                                 stride_width,
                                                 padding_depth,
                                                 padding_height,
                                                 padding_width,
                                                 adaptive,
                                                 input_grad_data,
                                                 pool_divmods_input,
                                                 pool_divmods_stride);
  }
};
*/

 /*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
 // SUB:DONE pytorch版的maxpool3d起反向
 template <typename T1, typename T2>
 class MaxPool3dWithIndexGradFunctor<phi::GPUContext, T1, T2> {
  public:
   void operator()(const phi::GPUContext& context,
                   const DenseTensor& output_grad,
                   const DenseTensor& mask,
                   const std::vector<int>& ksize,
                   const std::vector<int>& strides,
                   const std::vector<int>& paddings,
                   bool adaptive,
                   DenseTensor* input_grad) {
     const int batch_size = input_grad->dims()[0];
     const int input_channels = input_grad->dims()[1];
     const int input_depth = input_grad->dims()[2];
     const int input_height = input_grad->dims()[3];
     const int input_width = input_grad->dims()[4];
     const int output_depth = output_grad.dims()[2];
     const int output_height = output_grad.dims()[3];
     const int output_width = output_grad.dims()[4];
     const int ksize_depth = ksize[0];
     const int ksize_height = ksize[1];
     const int ksize_width = ksize[2];
     const int stride_depth = strides[0];
     const int stride_height = strides[1];
     const int stride_width = strides[2];
     const int padding_depth = paddings[0];
     const int padding_height = paddings[1];
     const int padding_width = paddings[2];
 
     const T1* output_grad_data = output_grad.data<T1>();
     const T2* mask_data = mask.data<T2>();
     T1* input_grad_data = context.template Alloc<T1>(input_grad);
 
     int ncd = batch_size * input_channels * output_depth;
 
    //  backends::gpu::GpuLaunchConfig config = backends::gpu::GetGpuLaunchConfig3D(context, ncd, output_height, output_width);
    //  dim3 threads = config.thread_per_block;
    //  dim3 grid = config.block_per_grid;
 
     int thread_x = 32;
     int thread_y = 8;
     int thread_z = 1;
     dim3 threads(thread_x, thread_y, thread_z);
     std::array<int, 3> max_grid_dim = context.GetCUDAMaxGridDimSize();
     int block_x = (output_width + threads.x - 1) / threads.x;
     int block_y = (output_height + threads.y - 1) / threads.y;
     int block_z = (ncd > max_grid_dim[2] * threads.z) ? max_grid_dim[2] : (ncd + threads.z - 1) / threads.z;
     dim3 grid(block_x, block_y, block_z);
       
     auto pool_divmods_output = FastDivModForPooling3D(input_channels, output_width, output_height, output_depth);
 
     KernelMaxPool3DWithIdxGrad<T1, T2>
         <<<grid, threads, 0, context.stream()>>>(ncd,
                                                  output_grad_data,
                                                  mask_data,
                                                  input_channels,
                                                  input_depth,
                                                  input_height,
                                                  input_width,
                                                  output_depth,
                                                  output_height,
                                                  output_width,
                                                  ksize_depth,
                                                  ksize_height,
                                                  ksize_width,
                                                  stride_depth,
                                                  stride_height,
                                                  stride_width,
                                                  padding_depth,
                                                  padding_height,
                                                  padding_width,
                                                  adaptive,
                                                  input_grad_data,
                                                  pool_divmods_output);
   }
 };

template class MaxPool3dWithIndexFunctor<phi::GPUContext, float, int>;
template class MaxPool3dWithIndexGradFunctor<phi::GPUContext, float, int>;
template class MaxPool3dWithIndexFunctor<phi::GPUContext, double, int>;
template class MaxPool3dWithIndexGradFunctor<phi::GPUContext, double, int>;

}  // namespace funcs
}  // namespace phi
