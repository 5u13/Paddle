#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/index_sample_kernel.h"

#include <algorithm>
#include <vector>

#include "paddle/fluid/framework/convert_utils.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

namespace {
#define PREDEFINED_BLOCK_SIZE_X 512
#define PREDEFINED_BLOCK_SIZE 1024
#define MIN(a, b) ((a) < (b) ? (a) : (b))
}  // namespace

// SUB:REF:TODO 参考它是怎么二维起kernel的
template <typename T, typename IndexT = int>
__global__ void IndexSampleForward(const IndexT* index,
                                   const T* in_data,
                                   T* out_data,
                                   size_t index_length,
                                   size_t input_length,
                                   size_t batch_size) {
  unsigned int index_i = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int index_j = blockDim.y * blockIdx.y + threadIdx.y;
  for (; index_j < batch_size; index_j += blockDim.y * gridDim.y) {
    index_i = blockDim.x * blockIdx.x + threadIdx.x;
    for (; index_i < index_length; index_i += blockDim.x * gridDim.x) {
      unsigned int index_idx = index_j * index_length + index_i;
      unsigned int in_idx = index_j * input_length + index_i;
      IndexT sample_idx = index[index_idx];
      out_data[index_idx] = in_data[in_idx - index_i + sample_idx];
    }
  }
}

// SUB:REF:TODO 参考它是怎么二维起kernel的
template <typename T, typename Context>
void IndexSampleKernel(const Context& ctx,
                       const DenseTensor& x,
                       const DenseTensor& index,
                       DenseTensor* out) {
  auto index_type = index.dtype();
  bool index_type_match =
      index_type == DataType::INT32 || index_type == DataType::INT64;
  PADDLE_ENFORCE_EQ(
      index_type_match,
      true,
      errors::InvalidArgument(
          "Input(Index) holds the wrong type, it holds %s, but "
          "desires to be %s or %s",
          paddle::framework::DataTypeToString(
              paddle::framework::TransToProtoVarType(index_type)),
          paddle::framework::DataTypeToString(
              paddle::framework::TransToProtoVarType(DataType::INT32)),
          paddle::framework::DataTypeToString(
              paddle::framework::TransToProtoVarType((DataType::INT64)))));
  const T* in_data = x.data<T>();
  T* out_data = ctx.template Alloc<T>(out);
  // 这是干嘛的？GPU单独一个stream，可是还有什么stream吗
  auto stream = reinterpret_cast<const phi::GPUContext&>(ctx).stream();
  auto input_dim = x.dims();
  auto index_dim = index.dims();
  size_t batch_size = input_dim[0];
  size_t input_length = input_dim[1];
  size_t index_length = index_dim[1];

  auto block_width = paddle::platform::RoundToPowerOfTwo(index_length);
  // 二维配置时，单维度最高不超过512
  // blocksize不宜过大？通常而言，越大越好，这样单个block执行的计算量大，减少创建block的开销，同时其实也不会减损occupancy，因为本质上是shm和reg在bound occupancy，blocksize增加其实也是同步增加occupancy的，不过各代gpu最大其实就2048
  block_width = MIN(block_width, PREDEFINED_BLOCK_SIZE_X);
  int block_height =
      paddle::platform::RoundToPowerOfTwo(index_length * batch_size) /
      block_width;
  // 总之就是把blocksize限制到1024以内，这个确实是对的，因为每个sm至少要起到2个block，而且软件block最多就起1024个
  block_height = MIN(block_height, PREDEFINED_BLOCK_SIZE / block_width);
  dim3 block_dim(block_width, block_height);
  dim3 grid_dim((index_length + block_dim.x - 1) / block_dim.x,
                (batch_size + block_dim.y - 1) / block_dim.y);
  // 这是干嘛的？就是不超过CUDA最高的gridsize
  paddle::platform::LimitGridDim(ctx, &grid_dim);

  if (index_type == DataType::INT64) {
    const int64_t* index_data = index.data<int64_t>();
    IndexSampleForward<T, int64_t><<<grid_dim, block_dim, 0, stream>>>(
        index_data, in_data, out_data, index_length, input_length, batch_size);
  } else if (index_type == DataType::INT32) {
    const int* index_data = index.data<int>();
    // 第三个参数shm size，第四个参数stream
    // 里面的写法也就y维度batch_size，x维度index_length
    // 二维的写法到底好在哪？二维block的局部性？相当于是blocking
    IndexSampleForward<T, int><<<grid_dim, block_dim, 0, stream>>>(
        index_data, in_data, out_data, index_length, input_length, batch_size);
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(index_sample,
                   GPU,
                   ALL_LAYOUT,
                   phi::IndexSampleKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
